#include "hip/hip_runtime.h"
/*
 * MicroHH
 * Copyright (c) 2011-2020 Chiel van Heerwaarden
 * Copyright (c) 2011-2020 Thijs Heus
 * Copyright (c) 2014-2020 Bart van Stratum
 *
 * This file is part of MicroHH
 *
 * MicroHH is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.

 * MicroHH is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.

 * You should have received a copy of the GNU General Public License
 * along with MicroHH.  If not, see <http://www.gnu.org/licenses/>.
 */

#include "canopy.h"
#include "grid.h"
#include "fast_math.h"
#include "fields.h"
#include "tools.h"

namespace
{
    namespace fm = Fast_math;

    template<typename TF> __global__
    void canopy_drag_u_g(
            TF* const restrict ut,
            const TF* const restrict u,
            const TF* const restrict v,
            const TF* const restrict w,
            const TF* const restrict pad,
            const TF utrans,
            const TF vtrans,
            const TF cd,
            const int istart, const int iend,
            const int jstart, const int jend,
            const int kstart, const int kend,
            const int jstride, const int kstride)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
        const int k = blockIdx.z + kstart;

        const int ii = 1;
        const int jj = jstride;
        const int kk = kstride;

        if (i < iend && j < jend && k < kend)
        {
            const int ijk = i + j*jj + k*kk;

            // Interpolate `v` and `w` to `u` locations.
            const TF u_on_u = u[ijk] + utrans;
            const TF v_on_u = TF(0.25) * (v[ijk] + v[ijk-ii] + v[ijk-ii+jj] + v[ijk+jj]) + vtrans;
            const TF w_on_u = TF(0.25) * (w[ijk] + w[ijk-ii] + w[ijk-ii+kk] + w[ijk+kk]);

            const TF ftau = -cd * pad[k] *
                pow( fm::pow2(u_on_u) +
                     fm::pow2(v_on_u) +
                     fm::pow2(w_on_u), TF(0.5) );

            ut[ijk] += ftau * u_on_u;
        }
    }

    template<typename TF> __global__
    void canopy_drag_v_g(
            TF* const restrict vt,
            const TF* const restrict u,
            const TF* const restrict v,
            const TF* const restrict w,
            const TF* const restrict pad,
            const TF utrans,
            const TF vtrans,
            const TF cd,
            const int istart, const int iend,
            const int jstart, const int jend,
            const int kstart, const int kend,
            const int jstride, const int kstride)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
        const int k = blockIdx.z + kstart;

        const int ii = 1;
        const int jj = jstride;
        const int kk = kstride;

        if (i < iend && j < jend && k < kend)
        {
            const int ijk = i + j*jj + k*kk;

            // Interpolate `u` and `w` to `v` locations.
            const TF u_on_v = TF(0.25) * (u[ijk] + u[ijk+ii] + u[ijk+ii-jj] + v[ijk-jj]) + utrans;
            const TF v_on_v = v[ijk] + vtrans;
            const TF w_on_v = TF(0.25) * (w[ijk] + w[ijk+kk] + w[ijk+kk-kk] + w[ijk-jj]);

            const TF ftau = -cd * pad[k] *
                pow( fm::pow2(u_on_v) +
                     fm::pow2(v_on_v) +
                     fm::pow2(w_on_v), TF(0.5) );

            vt[ijk] += ftau * v_on_v;
        }
    }

    template<typename TF> __global__
    void canopy_drag_w_g(
            TF* const restrict wt,
            const TF* const restrict u,
            const TF* const restrict v,
            const TF* const restrict w,
            const TF* const restrict padh,
            const TF utrans,
            const TF vtrans,
            const TF cd,
            const int istart, const int iend,
            const int jstart, const int jend,
            const int kstart, const int kend,
            const int jstride, const int kstride)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
        const int k = blockIdx.z + kstart+1;

        const int ii = 1;
        const int jj = jstride;
        const int kk = kstride;

        if (i < iend && j < jend && k < kend)
        {
            const int ijk = i + j*jj + k*kk;

            // Interpolate `u` and `v` to `w` locations.
            const TF u_on_w = TF(0.25) * (u[ijk] + u[ijk+ii] + u[ijk+ii-kk] + v[ijk-kk]) + utrans;
            const TF v_on_w = TF(0.25) * (v[ijk] + v[ijk+jj] + v[ijk+jj-kk] + v[ijk-kk]) + vtrans;
            const TF w_on_w = w[ijk];

            const TF ftau = -cd * padh[k] *
                pow( fm::pow2(u_on_w) +
                     fm::pow2(v_on_w) +
                     fm::pow2(w_on_w), TF(0.5) );

            wt[ijk] += ftau * w_on_w;
        }
    }
}

#ifdef USECUDA
template <typename TF>
void Canopy<TF>::prepare_device()
{
    if (!sw_canopy)
        return;

    auto& gd = grid.get_grid_data();
    const int nmemsize = gd.kcells*sizeof(TF);

    cuda_safe_call(hipMalloc(&pad_g,  nmemsize));
    cuda_safe_call(hipMalloc(&padh_g, nmemsize));

    cuda_safe_call(hipMemcpy(pad_g,  pad.data(),  nmemsize, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(padh_g, padh.data(), nmemsize, hipMemcpyHostToDevice));
}


template <typename TF>
void Canopy<TF>::clear_device()
{
    if (!sw_canopy)
        return;

    cuda_safe_call(hipFree(pad_g));
    cuda_safe_call(hipFree(padh_g));
}


template <typename TF>
void Canopy<TF>::exec()
{
    if (!sw_canopy)
        return;

    auto& gd = grid.get_grid_data();

    const int blocki = gd.ithread_block;
    const int blockj = gd.jthread_block;

    const int gridi  = gd.imax/blocki + (gd.imax%blocki > 0);
    const int gridj  = gd.jmax/blockj + (gd.jmax%blockj > 0);

    dim3 gridGPU (gridi, gridj, kend_canopy-gd.kstart);
    dim3 blockGPU(blocki, blockj, 1);

    // Momentum drag
    canopy_drag_u_g<TF><<<gridGPU, blockGPU>>>(
        fields.mt.at("u")->fld_g,
        fields.mp.at("u")->fld_g,
        fields.mp.at("v")->fld_g,
        fields.mp.at("w")->fld_g,
        pad_g,
        grid.utrans,
        grid.vtrans,
        cd,
        gd.istart, gd.iend,
        gd.jstart, gd.jend,
        gd.kstart, kend_canopy,
        gd.icells, gd.ijcells);
    cuda_check_error();

    canopy_drag_v_g<TF><<<gridGPU, blockGPU>>>(
        fields.mt.at("v")->fld_g,
        fields.mp.at("u")->fld_g,
        fields.mp.at("v")->fld_g,
        fields.mp.at("w")->fld_g,
        pad_g,
        grid.utrans,
        grid.vtrans,
        cd,
        gd.istart, gd.iend,
        gd.jstart, gd.jend,
        gd.kstart, kend_canopy,
        gd.icells, gd.ijcells);
    cuda_check_error();

    canopy_drag_w_g<TF><<<gridGPU, blockGPU>>>(
        fields.mt.at("w")->fld_g,
        fields.mp.at("u")->fld_g,
        fields.mp.at("v")->fld_g,
        fields.mp.at("w")->fld_g,
        padh_g,
        grid.utrans,
        grid.vtrans,
        cd,
        gd.istart, gd.iend,
        gd.jstart, gd.jend,
        gd.kstart, kend_canopy,
        gd.icells, gd.ijcells);
    cuda_check_error();
}
#endif

template class Canopy<double>;
template class Canopy<float>;
